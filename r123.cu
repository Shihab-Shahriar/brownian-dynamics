#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <sstream>
#include <vector>
#include <hip/hip_runtime.h>

#include "Random123/philox.h"
#include "Random123/uniform.hpp"

#define FQUALIFIER __host__ __device__

#define PI           3.14159265358979323846 
#define GLOBAL_SEED 0x43

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func <<" "<<hipGetErrorString(result)<< "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

typedef r123::Philox4x32 RNG;


// Radius of particles
const double RADIUS = 1.0;
const int N = 1000000; // Number of particles
const double dt = 0.05; // Time step
const double T = 64.0; // Temperature
const double GAMMA = 1.0; // Drag coefficient
const double mass = 1.0; // Mass of particles
const int STEPS = 10000; // Number of simulation steps

//Sim Box parameters
const int windowWidth = 800;
const int windowHeight = 600;


struct Particle {
    double x = 0;
    double y = 0;
    double vx = 0;
    double vy = 0;

    int pid = 0;

    FQUALIFIER Particle(double x, double y) : x(x), y(y) 
    {

    }

    FQUALIFIER void update(double dx, double dy) {
        x += dx; 
        if(x < 0)
            x = 0;
        else if(x > windowWidth)
            x = windowWidth;

        y += dy;
        if(y < 0)
            y = 0;
        else if(y > windowHeight)
            y = windowHeight;
    }

};


__global__ void init_particles(Particle *particles, int counter){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N)
        return;

    Particle p = particles[i];
    p.pid = i;

    RNG rng;
    RNG::ctr_type c={{}};
    RNG::ukey_type uk={{}};
    uk[0] = p.pid;
    RNG::key_type k=uk;

    c[0] = counter; 
    c[1] = 0;
    RNG::ctr_type r = rng(c, k);

    uint64_t xu =
        (static_cast<uint64_t>(r[0]) << 32) | static_cast<uint64_t>(r[1]);
    uint64_t yu =
        (static_cast<uint64_t>(r[2]) << 32) | static_cast<uint64_t>(r[3]);
    auto x = r123::u01<double, uint64_t>(xu);
    auto y = r123::u01<double, uint64_t>(yu);
    p.update(x, y);

    // increment counter
    c[1]++;
    r = rng(c, k);

    xu = (static_cast<uint64_t>(r[0]) << 32) | static_cast<uint64_t>(r[1]);
    yu = (static_cast<uint64_t>(r[2]) << 32) | static_cast<uint64_t>(r[3]);
    
    p.vx = r123::u01<double, uint64_t>(xu) * 100 - 50.0;
    p.vy = r123::u01<double, uint64_t>(yu) * 100 - 50.0;

    particles[i] = p;
}


__global__ void apply_forces(Particle *particles, int counter, double sqrt_dt){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N)
        return;

    Particle p = particles[i];
    // Apply drag force
    p.vx -= GAMMA / mass * p.vx * dt;
    p.vy -= GAMMA / mass * p.vy * dt;

    // Apply random force
    RNG rng;
    RNG::ctr_type c={{}};
    RNG::ukey_type uk={{}};
    uk[0] = p.pid;
    RNG::key_type k=uk;

    c[0] = counter; 
    c[1] = 0;
    RNG::ctr_type r = rng(c, k);

    uint64_t xu =
        (static_cast<uint64_t>(r[0]) << 32) | static_cast<uint64_t>(r[1]);
    uint64_t yu =
        (static_cast<uint64_t>(r[2]) << 32) | static_cast<uint64_t>(r[3]);
    auto x = r123::u01<double, uint64_t>(xu);
    auto y = r123::u01<double, uint64_t>(yu);

    p.vx += (x  * 2.0 - 1.0) * sqrt_dt;
    p.vy += (y  * 2.0 - 1.0) * sqrt_dt;
    particles[i] = p;

}

__global__ void update_positions(Particle *particles){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N)
        return;
        
    Particle p = particles[i];

    // Check for collisions with box boundaries
    if (p.x - RADIUS < 0 || p.x + RADIUS > windowWidth) {
        p.vx *= -1;
    }
    if (p.y - RADIUS < 0 || p.y + RADIUS > windowHeight) {
        p.vy *= -1;
    }
    // Update positions
    p.update(p.vx * dt, p.vy * dt);

    particles[i] = p;

}


int main(){
    const double sqrt_dt = std::sqrt(2.0 * T * GAMMA / mass * dt); // Standard deviation for random force
    std::cout << "sqrt_dt: " << sqrt_dt << "\n";

    const double density = (N * PI * RADIUS* RADIUS) / (windowWidth * windowHeight);
    std::cout << "density: " << density << "\n";


    // allocate particles
    Particle *particles;
    checkCudaErrors(hipMallocManaged((void **)&particles, N * sizeof(Particle)));

    const int nthreads = 256;
    const int nblocks = (N + nthreads - 1) / nthreads;

    // Initialize particles
    init_particles<<<nblocks, nthreads>>>(particles, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // Simulation loop
    int iter = 0;
    while (iter++ < STEPS) {
        apply_forces<<<nblocks, nthreads>>>(particles, iter, sqrt_dt);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        update_positions<<<nblocks, nthreads>>>(particles);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    }

    // Reproducibility check: output the positions. Turn off for benchmarking
//     for(int i=0; i<N; i++){
//         std::cout << particles[i].x << " " << particles[i].y << "\n";
//     }

    hipFree(particles);
    return 0;
}