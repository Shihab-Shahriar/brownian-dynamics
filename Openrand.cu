#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <sstream>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>

#include "phillox.h"

#define FQUALIFIER __host__ __device__

#define PI           3.14159265358979323846 
#define GLOBAL_SEED 0x43

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func <<" "<<hipGetErrorString(result)<< "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

typedef Phillox RNG;


// Radius of particles
const double RADIUS = 1.0;
const int N = 1000000; // Number of particles
const double dt = 0.05; // Time step
const double T = 64.0; // Temperature
const double GAMMA = 1.0; // Drag coefficient
const double mass = 1.0; // Mass of particles
const int STEPS = 10000; // Number of simulation steps

//Sim Box parameters
const int windowWidth = 800;
const int windowHeight = 600;


struct Particle {
    double x = 0;
    double y = 0;
    double vx = 0;
    double vy = 0;

    int pid = 0;

    FQUALIFIER Particle(double x, double y) : x(x), y(y) 
    {

    }

    FQUALIFIER void update(double dx, double dy) {
        x += dx; 
        if(x < 0)
            x = 0;
        else if(x > windowWidth)
            x = windowWidth;

        y += dy;
        if(y < 0)
            y = 0;
        else if(y > windowHeight)
            y = windowHeight;
    }

};


__global__ void init_particles(Particle *particles, int counter){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N)
        return;

    Particle p = particles[i];
    p.pid = i;

    RNG local_rand_state(p.pid, counter, GLOBAL_SEED);

    auto x = local_rand_state.rand<double>() * double(windowWidth) - 1.0;
    auto y = local_rand_state.rand<double>() * double(windowHeight) - 1.0;
    p.update(x, y);

    p.vx = local_rand_state.rand<double>() * 100 - 50.0;
    p.vy = local_rand_state.rand<double>() * 100 - 50.0;

    particles[i] = p;
}


__global__ void apply_forces(Particle *particles, int counter, double sqrt_dt){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N)
        return;

    Particle p = particles[i];
    // Apply drag force
    p.vx -= GAMMA / mass * p.vx * dt;
    p.vy -= GAMMA / mass * p.vy * dt;

    // Simulate collision using linear spring forces
    const double k = 2.0; 

    for (auto j : {-1, 1}) {
        j = (i + N + j) % N;  
        Particle q = particles[j];
        double dx = p.x - q.x;
        double dy = p.y - q.y;
        double dist = sqrt(dx * dx + dy * dy);

        if (dist < 4 * RADIUS) {  
            double force = -k * (dist - 2 * RADIUS);  
            double force_x = force * dx / dist;  
            double force_y = force * dy / dist;

            p.vx += force_x * dt;  
            p.vy += force_y * dt;
        }
    }

    // Apply random force
    RNG local_rand_state(p.pid, counter);
    
    //double2 r = hiprand_uniform2_double(&local_rand_state); 
    auto x = local_rand_state.rand<double>();
    auto y = local_rand_state.rand<double>();
    p.vx += (x  * 2.0 - 1.0) * sqrt_dt;
    p.vy += (y  * 2.0 - 1.0) * sqrt_dt;
    particles[i] = p;

}

__global__ void update_positions(Particle *particles){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N)
        return;
        
    Particle p = particles[i];

    // Check for collisions with box boundaries
    if (p.x - RADIUS < 0 || p.x + RADIUS > windowWidth) {
        p.vx *= -1;
    }
    if (p.y - RADIUS < 0 || p.y + RADIUS > windowHeight) {
        p.vy *= -1;
    }
    // Update positions
    p.update(p.vx * dt, p.vy * dt);

    particles[i] = p;

}


void test(Particle *particles, int nthreads){
    const double sqrt_dt = std::sqrt(2.0 * T * GAMMA / mass * dt); // Standard deviation for random force
    std::cout << "sqrt_dt: " << sqrt_dt << "\n";

    const double density = (N * PI * RADIUS* RADIUS) / (windowWidth * windowHeight);
    std::cout << "density: " << density << "\n";


    const int nblocks = (N + nthreads - 1) / nthreads;

    // Initialize particles
    init_particles<<<nblocks, nthreads>>>(particles, 0);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());


    // Simulation loop
    int iter = 0;
    while (iter++ < STEPS) {
        apply_forces<<<nblocks, nthreads>>>(particles, iter, sqrt_dt);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        update_positions<<<nblocks, nthreads>>>(particles);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());
    }


}

int main(int argc, char **argv) {
    // allocate particles
    Particle *particles;
    checkCudaErrors(hipMallocManaged((void **)&particles, N * sizeof(Particle)));

    bool benchmark = false;

    if(benchmark){
        // benchmark
        test(particles, 256);
        return 0;
    }
    else{
        // Reproducibility check
        test(particles, 256);
        Particle *particles2;
        checkCudaErrors(hipMallocManaged((void **)&particles2, N * sizeof(Particle)));
        test(particles2, 512);

        // Reproducibility check
        std::vector<double> errors (N*2);
        for(int i=0; i<N; i++){
            errors[i*2] = particles[i].x - particles2[i].x;
            errors[i*2+1] = particles[i].y - particles2[i].y;
        }

        // To reduce floating point errors
        std::sort(errors.begin(), errors.end());
        
        // average error
        double avg = 0;
        for(int i=0; i<N; i++){
            avg += errors[i];
        }
        std::cout << "Total error: " << avg << "\n";
        hipFree(particles2);
    }

    hipFree(particles);
    return 0;
}